#include "hip/hip_runtime.h"
#include "cuda2.cuh"
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <string>
#include <iostream>
#include <complex>
#include "MyComplex.h"
#include ""
#include <hip/hip_complex.h>
#include <math.h>
using namespace std;
#define M_PI 3.1415926535897931
void CalcFFT(hipDoubleComplex* cuArr, int powerOfTwo, bool orientation);
void Blend(hipDoubleComplex* arr, int nx){

	hipDoubleComplex t;
	int i, j, k;
	int n1 = nx >> 1;

	for (i = 0, j = 0, k = n1; i < nx - 1; i++, j = j + k)
	{
		if (i < j)
		{
			t = arr[j];
			arr[j] = arr[i];
			arr[i] = t;
		}
		k = n1;
		while (k <= j)
		{
			j = j - k;
			k = k >> 1;
		}
	}


}
__global__ void FFT(hipDoubleComplex* array, int *ll1){

	
	hipDoubleComplex w, u, t;
	int ip;
	int idx = threadIdx.x;
	u = make_hipDoubleComplex(1.0, 0.0);
	w = make_hipDoubleComplex(cos(M_PI / (*ll1)), sin(M_PI / (*ll1)));
	if (idx >= *ll1){
		int mod = idx/(*ll1);
		idx = idx + (*ll1) * mod;
		int ctrl = idx - (*ll1) * mod * 2;
		for (int i = 0; i < ctrl; i++) {
			u = hipCmul(u, w);
		}
	}

	ip = idx + (*ll1);
	t = hipCmul(array[ip], u);
	array[ip] = hipCsub(array[idx], t);
	array[idx] = hipCadd(array[idx], t);

}
__global__ void iFFT(hipDoubleComplex* array, int *ll1){


	hipDoubleComplex w, u, t;
	int ip;
	int idx = threadIdx.x;
	u = make_hipDoubleComplex(1.0, 0.0);
	w = make_hipDoubleComplex(cos(M_PI / (*ll1)), sin(-M_PI / (*ll1)));
	if (idx >= *ll1){
		int mod = idx / (*ll1);
		idx = idx + (*ll1) * mod;
		int ctrl = idx - (*ll1) * mod * 2;
		for (int i = 0; i < ctrl; i++) {
			u = hipCmul(u, w);
		}
	}

	ip = idx + (*ll1);
	t = hipCmul(array[ip], u);
	array[ip] = hipCsub(array[idx], t);
	array[idx] = hipCadd(array[idx], t);


}

__global__ void DIV(hipDoubleComplex* array, int* powerOfTwo){

	double2 n;
	n.x = sqrt(pow(2.0,(double)(*powerOfTwo)));
	n.y = 0;
	int i = threadIdx.x;
	array[i] = hipCdiv(array[i], n);
}

void beforeFFT(MyComplex *cmplx, MyComplex *revcmplx, int powerOfTwo, bool orientation){

	int nx = cmplx->width;
	int ny = cmplx->height;
	hipDoubleComplex *cuArr = new hipDoubleComplex[nx];

	//�������
	for (int i = 0; i < nx; i++){
		for (int j = 0; j < ny; j++){
			cuArr[j] = make_hipDoubleComplex(cmplx->cmplx[j][i].real(), cmplx->cmplx[j][i].imag());
		}

		Blend(cuArr, nx);
		CalcFFT(cuArr,powerOfTwo,orientation);

		for (int j = 0; j < ny; j++){
			revcmplx->cmplx[j][i] = complex<double>(cuArr[j].x, cuArr[j].y);
		}

	}

	//������
	for (int i = 0; i < nx; i++){
		for (int j = 0; j < ny; j++){
			cuArr[j] = make_hipDoubleComplex(revcmplx->cmplx[i][j].real(), revcmplx->cmplx[i][j].imag());
		}

		Blend(cuArr, nx);
		CalcFFT(cuArr, powerOfTwo, orientation);

		for (int j = 0; j < ny; j++){
			revcmplx->cmplx[i][j] = complex<double>(cuArr[j].x, cuArr[j].y);
		}
	}

}


void CalcFFT(hipDoubleComplex* cuArr, int powerOfTwo, bool orientation){

	int nx = pow(2.0, powerOfTwo);
	int *devll1, *PoW;
	dim3 grids = dim3(1, 1, 1);
	dim3 blocks = dim3(nx >> 1, 1, 1);
	hipDoubleComplex *devcuArr;
	hipMalloc((hipDoubleComplex**)&devcuArr, sizeof(hipDoubleComplex)*nx);
	hipMalloc((int**)&PoW, sizeof(int));
	hipMemcpy(PoW, &powerOfTwo, sizeof(int), hipMemcpyHostToDevice);
	hipMalloc((int**)&devll1, sizeof(int));

		hipMemcpy(devcuArr, cuArr, sizeof(hipDoubleComplex)*nx, hipMemcpyHostToDevice);

		for (int i = 1; i <= powerOfTwo; i++){

			int ll = (pow(2.0, i));
			int ll1 = ll >> 1;
			hipMemcpy(devll1, &ll1, sizeof(int), hipMemcpyHostToDevice);
			if (orientation)
				FFT << <grids, blocks >> >(devcuArr, devll1); 
			else 
				iFFT << <grids, blocks >> >(devcuArr, devll1);

			hipEvent_t syncEvent;
			hipEventCreate(&syncEvent);    //������� event 
			hipEventRecord(syncEvent, 0);  //���������� event 
			hipEventSynchronize(syncEvent);  //�������������� event

		}
		DIV << <1, nx >> >(devcuArr, PoW);
		hipMemcpy(cuArr, devcuArr, sizeof(hipDoubleComplex)*nx, hipMemcpyDeviceToHost);

	hipFree(devcuArr);
}


void Start_Cuda(MyComplex cmplx, MyComplex revcmplx, bool or){

	int m = 1;
	int n = cmplx.width;
	int nn = 2;

	for (int i = 1;; i++) { nn = nn * 2; if (nn > n) { n = nn / 2; m = i; break; } }
	beforeFFT(&cmplx, &revcmplx, m, or);
	

}
std::string INFO(){
	std::string str;
	int devices;
	hipDeviceProp_t info;
	hipGetDeviceCount(&devices);

	str = "���������� GPU �������������� CUDA: ";
	str += std::to_string(devices);
	str += ";";

	for (int i = 0; i<devices; i++)
	{
		hipGetDeviceProperties(&info, i);
		str += "�������� GPU: ";
		str += info.name;
		str += ";";
		str += "��������� ������: ";
		str += std::to_string(info.totalGlobalMem / 1048576);
		str += " MB";
		str += ";";
		str += "��������� ���������� ������ ������: ";
		str += std::to_string(info.totalConstMem);
		str += " B";
		str += ";";
		str += "����� ������ ��� ������: ";
		str += std::to_string(info.sharedMemPerBlock);
		str += " B";
		str += ";";
		str += "����� ���������� 32 - ������ ���������: ";
		str += std::to_string(info.regsPerBlock);
		str += ";";
		str += "������ Warp: ";
		str += std::to_string(info.warpSize);
		str += ";";
		str += "������������ ���������� ������� � �����: ";
		str += std::to_string(info.maxThreadsPerBlock);
		str += ";";
		str += "������������ ������ �����: ";
		str += std::to_string(info.maxThreadsDim[0]);
		for (int i = 1; i < 3; i++){
			str += "x";
			str += std::to_string(info.maxThreadsDim[i]);

		}
		str += ";";
		str += "������������ ������ �����: ";
		str += std::to_string(info.maxGridSize[0]);
		for (int i = 1; i < 3; i++){
			str += "x";
			str += std::to_string(info.maxGridSize[i]);


		}
		str += ";";
		str += "�������� �������: ";
		str += std::to_string(info.clockRate / 1000);
		str += " MHz";
		str += ";";
		str += "������� ����: ";
		str += std::to_string(info.memoryClockRate / 1000);
		str += " MHz";
		str += ";";
		str += "������ ����: ";
		str += std::to_string(info.memoryBusWidth);
		str += ";";
		str += "��� l2: ";
		str += std::to_string(info.l2CacheSize);
		str += " B";
		str += ";";
	}
	return str;
}